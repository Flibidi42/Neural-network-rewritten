#include "hip/hip_runtime.h"
#include "../hd/class.hpp"

using namespace std;

Net::Net(int *size_layers, int nb_layers, int nb_input){
	
	m_nb_layers = nb_layers;
	m_nb_input = nb_input;
	for(int i = 0; i< nb_input; i++){
		m_size_layers[i] = size_layers[i];
	}
	
	//max_size for layers
	int max = 0;
	for(int i = 0; i<nb_layers; i++){
		if(size_layers[i] > max)
			max = size_layers[i];
	}
	m_max_lay = max;
	
	//bias - alloc
	m_bias = new float*[nb_layers];
	for(int i = 0; i<nb_layers; i++){
		m_bias[i] = new float[size_layers[i]];
	}
	//bias - init
	for(int i = 0; i<nb_layers; i++){
		for(int j = 0; j<size_layers[i]; j++){
			m_bias[i][j] = 0;
		}
	}
		
	//weight - alloc/init
	m_weight = new float**[nb_layers];
	for(int i = 0; i<nb_layers; i++){
		m_weight[i] = new float*[size_layers[i]];
	}
	for(int i = 0; i<nb_layers; i++){
		for(int j = 0; j<size_layers[i]; j++){
			if(i == 0){ // particular case : first layer
				m_weight[i][j] = new float[nb_input];
				for(int k = 0; k<nb_input; k++){ // init
					m_weight[i][j][k] = 1;
				}
			}
			else{ // other layers
				m_weight[i][j] = new float[size_layers[i-1]];
				for(int k = 0; k<size_layers[i-1]; k++){ // init
					m_weight[i][j][k] = 1;
				}
			}
		}
	}
	
	//grad - alloc/init
	m_grad = new float*[nb_layers];
	for(int i = 0; i<nb_layers; i++){
		m_grad[i] = new float[size_layers[i]];
	}
	//grad - init
	for(int i = 0; i<nb_layers; i++){
		for(int j = 0; j<size_layers[i]; j++){
			m_grad[i][j] = 0;
		}
	}
	
	//delta - alloc/init
	m_delta = new float**[nb_layers];
	for(int i = 0; i<nb_layers; i++){
		m_delta[i] = new float*[size_layers[i]];
	}
	for(int i = 0; i<nb_layers; i++){
		for(int j = 0; j<size_layers[i]; j++){
			if(i == 0){ // particular case : first layer
				m_delta[i][j] = new float[nb_input];
				for(int k = 0; k<nb_input; k++){ // init
					m_delta[i][j][k] = 0;
				}
			}
			else{ // other layers
				m_delta[i][j] = new float[size_layers[i-1]];
				for(int k = 0; k<size_layers[i-1]; k++){ // init
					m_delta[i][j][k] = 0;
				}
			}
		}
	}
}

void Net::learning(float* input, float *expect){
	
	float* output = comput(input);
	float error = 0;
	float** transition = new float*[m_nb_layers];
	for(int i = 0; i<m_nb_layers; i++){
		transition[i] = new float[m_max_lay];
		for(int j = 0; j<m_max_lay; j++){
			transition[i][j] = 0;
		}
	}
	
	//computation with memory
	
	for(int i = 0; i<m_nb_layers; i++){ // for each layer
		for(int j = 0; j<m_size_layers[i]; j++){ // for each neuron
			//calcul layers
			if(i == 0){
				for(int k = 0; k<m_nb_input; k++){ // sum
					transition[0][j] += input[k] * m_weight[i][j][k];
				}
				transition[0][j] += m_bias[i][j];//bias
				transition[0][j] = sigmo(transition[0][j]);
			}
			else{
				for(int k = 0; k<m_size_layers[i-1]; k++){ // sum
					transition[i][j] += transition[i-1][k] * m_weight[i][j][k];
				}
				transition[i][j] += m_bias[i][j];//bias
				transition[i][j] = sigmo(transition[i][j]);
			}
		}
	}
	
	//error computation

	error = comput_error(transition[m_nb_layers-1], expect);
	
	//backprop
	
	float derivation_factor = 1.f;
	float y*;
	
	for(int i = m_nb_layers-1; i>=0; i--){ // for each layer
		for(int j = 0; j<m_size_layers[i]; j++){ // for each neuron
		
			y = transition[i];
			
			//grad / delta
			if(i == m_nb_layers-1){ // output neuron
			
				grad[i][j] = - (expect[j]-y[j]) * y[j] * (1-y[j]);
				for(int k= 0; k<m_size_layers[i-1]; k++){
					delta[i][j][k] = grad[i][j] * transition[i-1][k];
				}
			}
			
			else if(i  == 0){ // input neuron
				for(int k= 0; k<m_size_layers[i+1]; k++){
					grad[i][j] = grad[i+1][k] * weight[i+1][j][k] * y[j] * (1-y[j]);
				}
				for(int k= 0; k<m_size_layers[i-1]; k++){
					delta[i][j][k] = grad[i][j] * input[k];
				}
			}
			
			else{ // others
				for(int k= 0; k<m_size_layers[i+1]; k++){
					grad[i][j] = grad[i+1][k] * weight[i+1][j][k] * y[j] * (1-y[j]);
				}
				for(int k= 0; k<m_size_layers[i-1]; k++){
					delta[i][j][k] = grad[i][j] * transition[i-1][k];
				}
			}
			
		}
	}
	
}

void Net::backprop(){
	
	
	
}

float Net::comput_error(float* out, float* expect){
	
	float error = 0;
	
	for(int i = 0; i<m_size_layers[m_nb_layers-1]; i++){
		error += 0.5*(out[i] - expect[i])*(out[i] - expect[i]);
	}
	return error;	
}

float* Net::comput(float* input){
	
	float* transition = new float[m_max_lay];
	float* transition_old = new float[m_max_lay];
	
	for(int i = 0; i<m_nb_layers; i++){ // for each layer
		for(int j = 0; j<m_size_layers[i]; j++){ // for each neuron
			transition[j] = 0;
			//calcul layers
			if(i == 0){
				for(int k = 0; k<m_nb_input; k++){ // sum
					transition[j] += input[k] * m_weight[i][j][k];
				}
				transition[j] += m_bias[i][j];//bias
				transition[j] = sigmo(transition[j]);
			}
			else{
				for(int k = 0; k<m_size_layers[i-1]; k++){ // sum
					transition[j] += transition_old[k] * m_weight[i][j][k];
				}
				transition[j] += m_bias[i][j];//bias
				transition[j] = sigmo(transition[j]);
			}
		}
		//switch transistion
		for(int k = 0; k<m_size_layers[i]; k++){
				transition_old[k] = transition[k];
		}
	}
	return transition;
	
}

float Net::sigmo(float val){
    return 1/(1+exp(-val));
}