#include "hip/hip_runtime.h"
#include "../hd/class.hpp"

Net::Net(int *size_layers, int nb_layers, int nb_input){
	
	m_nb_layers = nb_layers;
	m_nb_input = nb_input;
	for(int i = 0; i< nb_input; i++){
		m_size_layer[i] = size_layer[i];
	}
	
	//max_size for layers
	int max = 0;
	for(int i = 0; i<nb_layers; i++){
		if(size_layers[i] > max)
			max = size_layers[i];
	}
	m_max_lay = max;
	
	//bias - alloc
	m_bias = new float*[nb_layers];
	for(int i = 0; i<nb_layers; i++){
		m_bias[i] = new float[size_layers[i]];
	}
	//bias - init
	for(int i = 0; i<nb_layers; i++){
		for(int j = 0; j<size_layers[i]; j++){
			m_bias[i][j] = 0.5;
		}
	}
		
	//weight - alloc/init
	m_weight = new float**[nb_layers];
	for(int i = 0; i<nb_layers; i++){
		m_weight[i] = new float*[size_layer[i]];
	}
	for(int i = 0; i<nb_layers; i++){
		for(int j = 0; j<size_layers[i]; j++){
			if(i == 0){ // particular case : first layer
				m_weight[i][j] = new float[nb_input];
				for(int k = 0; k<nb_input; k++){ // init
					m_weight[i][j][k] = 0.5;
				}
			}
			else{ // other layers
				m_weight[i][j] = new float[size_layers[i-1]];
				for(int k = 0; k<size_layers[i-1]; k++){ // init
					m_weight[i][j][k] = 0.5;
				}
			}
		}
	}
	
	//grad - alloc/init
	m_grad = new float*[nb_layers];
	for(int i = 0; i<nb_layers; i++){
		m_grad[i] = new float[size_layers[i]];
	}
	//grad - init
	for(int i = 0; i<nb_layers; i++){
		for(int j = 0; j<size_layers[i]; j++){
			m_grad[i][j] = 0;
		}
	}
	
	//delta - alloc/init
	m_delta = new float**[nb_layers];
	for(int i = 0; i<nb_layers; i++){
		m_delta[i] = new float*[size_layer[i]];
	}
	for(int i = 0; i<nb_layers; i++){
		for(int j = 0; j<size_layers[i]; j++){
			if(i == 0){ // particular case : first layer
				m_delta[i][j] = new float[nb_input];
				for(int k = 0; k<nb_input; k++){ // init
					m_delta[i][j][k] = 0;
				}
			}
			else{ // other layers
				m_delta[i][j] = new float[size_layers[i-1]];
				for(int k = 0; k<size_layers[i-1]; k++){ // init
					m_delta[i][j][k] = 0;
				}
			}
		}
	}
	
}